﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


int main()
{
	/*
		Cuda toolkit 1.0 versiyonu 2007 tarihinde NVIDA tarafindan yayimlanmistir.

		NVIDIA grafik kartlarinin cuda compute capability seviyeleri vardir.
		Bu seviyelere gore cihazin sahip oldugu temel ozelliklerde farklilasmalar
		vardir. Oregin bir grid icinde her yon(x, y, z) icin izin verilen maksimum
		thread block sayisi cihazdan cihaza farklilik gostermektedir.

		Gelistirdigimiz bir cuda uygulamasi farkli cuda capability seviyesine
		sahip cihazlar uzerinde calisacak ise cihazin ozelliklerini sorgulayabilmemiz
		gerekmektedir. Bu sayede farkli cihazlarda da uygulamamizin calisabilir hale
		getirebiliriz.
	
	*/

	int deviceCount = 0;

	hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0)
	{
		printf("No CUDA support device found");
		return;
	}
	printf("Number of devices: %d\n", deviceCount);
	int devNo = 0;
	hipDeviceProp_t iProp;
	hipGetDeviceProperties(&iProp, devNo);

	printf("Device %d: %s\n", devNo, iProp.name);
	printf("	Number of multiprocessros:	%d\n", iProp.multiProcessorCount);
	printf("	Clock rate:	%d\n", iProp.clockRate);
	printf("	Compute Capability:	%d.%d\n", iProp.major, iProp.minor);
	printf("	Total amount of global memory:	%4.2f KB\n",
		iProp.totalGlobalMem / 1024.0);
	printf("	Total amount of constant memory:	%4.2f KB\n",
		iProp.totalConstMem / 1024.0);
	printf("	Total amount of shared memory per block:	%4.2f KB\n",
		iProp.sharedMemPerBlock / 1024.0);
	printf("	Total amount of shared memory per MP:	%4.2f KB\n",
		iProp.sharedMemPerMultiprocessor / 1024.0);
	printf("	Total number of registers available per block:	%d\n",
		iProp.regsPerBlock);
	printf("	Warp size:	%d\n", iProp.warpSize);
	printf("	Maximum number of threads per block:	%d\n",
		iProp.maxThreadsPerBlock);
	printf("	Maximum number of threads per multiprocessor:	%d\n",
		iProp.maxThreadsPerMultiProcessor);
	printf("	Maximum grid size:	(%d, %d, %d)\n",
		iProp.maxGridSize[0], iProp.maxGridSize[1], iProp.maxGridSize[2]);
	printf("	Maximum block dimension:	(%d, %d, %d)\n",
		iProp.maxThreadsDim[0], iProp.maxThreadsDim[1], iProp.maxThreadsDim[2]);


}

